#include "hip/hip_runtime.h"
#include "ModulesCUDA.h"

#if COMPILECUDA == 1

#include "BorisCUDALib.cuh"

__global__ void ZeroEnergy_kernel(cuBReal& energy, cuReal3& torque, size_t& points_count)
{
	if (threadIdx.x == 0) energy = 0.0;
	if (threadIdx.x == 1) torque = 0.0;
	if (threadIdx.x == 2) points_count = 0;
}

void ModulesCUDA::ZeroEnergy(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		ZeroEnergy_kernel <<< 1, CUDATHREADS >>> (energy(mGPU), torque(mGPU), points_count(mGPU));
	}
}

__global__ void ZeroModuleVECs_kernel(cuVEC<cuReal3>& Module_Heff, cuVEC<cuReal3>& Module_Heff2, cuVEC<cuBReal>& Module_energy, cuVEC<cuBReal>& Module_energy2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Module_Heff.linear_size()) Module_Heff[idx] = cuReal3();
	if (idx < Module_Heff2.linear_size()) Module_Heff2[idx] = cuReal3();
	if (idx < Module_energy.linear_size()) Module_energy[idx] = 0.0;
	if (idx < Module_energy2.linear_size()) Module_energy2[idx] = 0.0;
}

void ModulesCUDA::ZeroModuleVECs(void)
{
	//This method is used at runtime, so better use a single kernel launch rather than zeroing them separately

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		ZeroModuleVECs_kernel <<< (Module_Heff.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(Module_Heff.get_deviceobject(mGPU), Module_Heff2.get_deviceobject(mGPU), Module_energy.get_deviceobject(mGPU), Module_energy2.get_deviceobject(mGPU));
	}
}

//-------------------------- Effective field and energy VECs

//Make sure memory is allocated correctly for display data if used, else free memory
BError ModulesCUDA::Update_Module_Display_VECs(cuReal3 h, cuRect meshRect, bool Module_Heff_used, bool Module_Energy_used, bool twosublattice)
{
	BError error(CLASS_STR(ModulesCUDA));

	//1. Heff - sub-lattice A

	if (Module_Heff.size_cpu().dim()) {

		if (Module_Heff_used && !Module_Heff.resize(h, meshRect)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
		else if (!Module_Heff_used) Module_Heff.clear();	
	}
	else if (Module_Heff_used) {

		if (!Module_Heff.assign(h, meshRect, cuReal3())) return error(BERROR_OUTOFGPUMEMORY_CRIT);
	}

	//2. Heff - sub-lattice B

	if (Module_Heff2.size_cpu().dim()) {

		if (twosublattice && Module_Heff_used && !Module_Heff2.resize(h, meshRect)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
		else Module_Heff2.clear();
	}
	else if (twosublattice && Module_Heff_used) {

		if (!Module_Heff2.assign(h, meshRect, cuReal3())) return error(BERROR_OUTOFGPUMEMORY_CRIT);
	}

	//3. Energy Density - sub-lattice A

	if (Module_energy.size_cpu().dim()) {

		if (Module_Energy_used && !Module_energy.resize(h, meshRect)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
		else if (!Module_Energy_used) Module_energy.clear();
	}
	else if (Module_Energy_used) {

		if (!Module_energy.assign(h, meshRect, 0.0)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
	}

	//4. Energy Density - sub-lattice B

	if (Module_energy2.size_cpu().dim()) {

		if (twosublattice && Module_Energy_used && !Module_energy2.resize(h, meshRect)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
		else Module_energy2.clear();
	}
	else if (twosublattice && Module_Energy_used) {

		if (!Module_energy2.assign(h, meshRect, 0.0)) return error(BERROR_OUTOFGPUMEMORY_CRIT);
	}

	return error;
}

//-------------------------- Torque

__global__ void CalculateTorque_kernel(cuVEC_VC<cuReal3>& M, cuVEC<cuReal3>& Module_Heff, cuRect avRect, cuReal3& torque, size_t& points_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3& n = M.n;

	cuReal3 torque_ = cuReal3();
	bool include_in_average = false;

	if (idx < M.linear_size()) {

		cuINT3 ijk = cuINT3(idx % n.x, (idx / n.x) % n.y, idx / (n.x*n.y));

		if (M.box_from_rect_max(avRect + M.rect.s).contains(ijk) && M.is_not_empty(ijk)) {

			torque_ = M[ijk] ^ Module_Heff[ijk];
			include_in_average = true;
		}
	}

	//need the idx < n.dim() check before cuvec.is_not_empty(ijk) to avoid bad memory access
	reduction_avg(0, 1, &torque_, torque, points_count, include_in_average);
}

//return cross product of M with Module_Heff, averaged in given rect (relative)
cuReal3 ModulesCUDA::CalculateTorque(mcu_VEC_VC(cuReal3)& M, cuRect& avRect)
{
	if (!Module_Heff.linear_size_cpu()) return cuReal3();

	ZeroEnergy();

	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		CalculateTorque_kernel <<< (M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
			(M.get_deviceobject(mGPU), Module_Heff.get_deviceobject(mGPU), avRect, torque(mGPU), points_count(mGPU));
	}

	size_t points_count_cpu = points_count.to_cpu_sum();

	if (points_count_cpu) return torque.to_cpu_sum() / points_count_cpu;
	else return cuReal3();
}

#endif