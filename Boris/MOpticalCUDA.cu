#include "hip/hip_runtime.h"
#include "MOpticalCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_MOPTICAL

#include "BorisCUDALib.cuh"

#include "MeshDefs.h"
#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void MOpticalCUDA_UpdateField_FM(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuBReal cHmo = *cuMesh.pcHmo;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHmo, cHmo);

		//magneto-optical field along z direction only : spatial and time dependence set through the usual material parameter mechanism
		Heff[idx] += cuReal3(0, 0, cHmo);

		if (do_reduction) {

			int non_empty_cells = M.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MU0 * M[idx] * cuReal3(0, 0, cHmo) / non_empty_cells;
		}

		if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = cuReal3(0, 0, cHmo);
		if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -(cuBReal)MU0 * M[idx] * cuReal3(0, 0, cHmo);
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

__global__ void MOpticalCUDA_UpdateField_AFM(ManagedMeshCUDA& cuMesh, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuReal3>& M2 = *cuMesh.pM2;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;
	cuVEC<cuReal3>& Heff2 = *cuMesh.pHeff2;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuBReal energy_ = 0.0;

	if (idx < Heff.linear_size()) {

		cuBReal cHmo = *cuMesh.pcHmo;
		cuMesh.update_parameters_mcoarse(idx, *cuMesh.pcHmo, cHmo);

		//magneto-optical field along z direction only : spatial and time dependence set through the usual material parameter mechanism
		Heff[idx] += cuReal3(0, 0, cHmo);
		Heff2[idx] += cuReal3(0, 0, cHmo);

		if (do_reduction) {

			int non_empty_cells = M.get_nonempty_cells();
			if (non_empty_cells) energy_ = -(cuBReal)MU0 * (M[idx] + M2[idx]) * cuReal3(0, 0, cHmo) / (2 * non_empty_cells);
		}

		if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[idx] = cuReal3(0, 0, cHmo);
		if (do_reduction && cuModule.pModule_Heff2->linear_size()) (*cuModule.pModule_Heff2)[idx] = cuReal3(0, 0, cHmo);
		if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[idx] = -MU0 * M[idx] * cuReal3(0, 0, cHmo);
		if (do_reduction && cuModule.pModule_energy2->linear_size()) (*cuModule.pModule_energy2)[idx] = -MU0 * M2[idx] * cuReal3(0, 0, cHmo);
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void MOpticalCUDA::UpdateField(void)
{
	if (pMeshCUDA->GetMeshType() == MESH_ANTIFERROMAGNETIC) {

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MOpticalCUDA_UpdateField_AFM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MOpticalCUDA_UpdateField_AFM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}

	else {

		if (pMeshCUDA->CurrentTimeStepSolved()) {

			ZeroEnergy();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MOpticalCUDA_UpdateField_FM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
		else {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				MOpticalCUDA_UpdateField_FM <<< (pMeshCUDA->M.device_size(mGPU) + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>> 
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

#endif

#endif