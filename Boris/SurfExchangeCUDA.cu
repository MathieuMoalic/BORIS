#include "hip/hip_runtime.h"
#include "SurfExchangeCUDA.h"

#if COMPILECUDA == 1

#ifdef MODULE_COMPILATION_SURFEXCHANGE

#include "BorisCUDALib.cuh"

#include "Mesh_FerromagneticCUDA.h"
#include "MeshParamsControlCUDA.h"

#include "ManagedAtom_MeshCUDA.h"
#include "Atom_MeshParamsControlCUDA.h"

#include "MeshDefs.h"

//Top mesh is ferromagnetic
__global__ void SurfExchangeCUDA_TopFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = pMesh_Top[mesh_idx].pM->mcuvec();

				//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
					M_Top.h.z / 2);

				//can't couple to an empty cell
				if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

				cuBReal J1 = *(pMesh_Top[mesh_idx].pJ1);
				cuBReal J2 = *(pMesh_Top[mesh_idx].pJ2);
				pMesh_Top[mesh_idx].update_parameters_atposition(cell_rel_pos, *(pMesh_Top[mesh_idx].pJ1), J1, *(pMesh_Top[mesh_idx].pJ2), J2);

				//get magnetization value in top mesh cell to couple with
				cuReal3 m_j = cu_normalize(M_Top[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				//NOTE : we must add into the module display VECs, since there could be 2 contributions for some cells (top and bottom). This is why we had to zero the VECs before calling this kernel.
				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Top mesh is antiferromagnetic
__global__ void SurfExchangeCUDA_TopAFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int k = idx / (n.x * n.y);
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Top = pMesh_Top[mesh_idx].pM->mcuvec();
				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Top = pMesh_Top[mesh_idx].pM2->mcuvec();

				//relative coordinates to read value from top mesh (the one we're coupling to here) - relative to top mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Top.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Top.rect.s.y,
					M_Top.h.z / 2);

				//can't couple to an empty cell
				if (!M_Top.rect.contains(cell_rel_pos + M_Top.rect.s) || M_Top.is_empty(cell_rel_pos)) continue;

				cuBReal J1 = *(pMesh_Top[mesh_idx].pJ1);
				cuBReal J2 = *(pMesh_Top[mesh_idx].pJ2);
				pMesh_Top[mesh_idx].update_parameters_atposition(cell_rel_pos, *(pMesh_Top[mesh_idx].pJ1), J1, *(pMesh_Top[mesh_idx].pJ2), J2);

				//get magnetization value in top mesh cell to couple with
				cuReal3 m_j1 = cu_normalize(M_Top[cell_rel_pos]);
				cuReal3 m_j2 = cu_normalize(M2_Top[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				//total surface exchange field in coupling cells, including contributions from both sub-lattices
				cuReal3 Hsurfexch = (m_j1 / ((cuBReal)MU0 * Ms * h.z)) * J1 + (m_j2 / ((cuBReal)MU0 * Ms * h.z)) * J2;

				if (do_reduction) {

					energy_ = (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Top mesh is atomistic
__global__ void SurfExchangeCUDA_TopAtom_UpdateField(ManagedMeshCUDA& cuMesh, ManagedAtom_MeshCUDA* pMesh_Top, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x + (n.z - 1) * n.x * n.y;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = pMesh_Top[mesh_idx].pM1->mcuvec();

				//coupling rectangle in atomistic mesh in absolute coordinates
				cuRect rect_c = cuRect(
					cuReal3(i * h.x, j * h.y, M.rect.e.z),
					cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.h.z + M.rect.e.z));
				rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

				//cells box in atomistic mesh
				cuBox acells = M1.box_from_rect_min(rect_c);

				//find total "directed energy" contribution from atomistic mesh : i.e. sum all mj * Js contributions from atomistic moments in the coupling area at the interface
				cuReal3 total_directed_coupling_energy = cuReal3();
				for (int ai = acells.s.i; ai < acells.e.i; ai++) {
					for (int aj = acells.s.j; aj < acells.e.j; aj++) {

						cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, M1.h.z / 2);

						if (M1.is_empty(rel_pos)) continue;

						//Js value from atomistic mesh
						cuBReal Js = *pMesh_Top[mesh_idx].pJs;
						cuBReal mu_s = *pMesh_Top[mesh_idx].pmu_s;
						pMesh_Top[mesh_idx].update_parameters_atposition(rel_pos, *pMesh_Top[mesh_idx].pJs, Js, *pMesh_Top[mesh_idx].pmu_s, mu_s);

						total_directed_coupling_energy += M1[rel_pos] * Js / mu_s;
					}
				}

				//now obtain coupling field from atomistic mesh at micromagnetic cell
				cuReal3 Hsurfexch = (total_directed_coupling_energy / (h.x * h.y)) / (MU0 * Ms * h.z);

				if (do_reduction) {

					energy_ = -(cuBReal)MU0 * M[cell_idx] * Hsurfexch / M.get_nonempty_cells();
				}

				Heff[cell_idx] += Hsurfexch;

				//NOTE : we must add into the module display VECs, since there could be 2 contributions for some cells (top and bottom). This is why we had to zero the VECs before calling this kernel.
				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is ferromagnetic
__global__ void SurfExchangeCUDA_BotFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);
			
			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = pMesh_Bot[mesh_idx].pM->mcuvec();

				//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
					M_Bot.rect.height() - M_Bot.h.z / 2);

				//can't couple to an empty cell
				if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

				//yes, then get value of magnetization used in coupling with current cell at cell_idx
				cuReal3 m_j = cu_normalize(M_Bot[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				cuBReal dot_prod = m_i * m_j;

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j / ((cuBReal)MU0 * Ms * h.z)) * (J1 + 2 * J2 * dot_prod);

				if (do_reduction) {

					energy_ = (-1 * J1 - 2 * J2 * dot_prod) * dot_prod / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is antiferromagnetic
__global__ void SurfExchangeCUDA_BotAFM_UpdateField(ManagedMeshCUDA& cuMesh, ManagedMeshCUDA* pMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int k = idx / (n.x * n.y);
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuBReal J1 = *cuMesh.pJ1;
			cuBReal J2 = *cuMesh.pJ2;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms, *cuMesh.pJ1, J1, *cuMesh.pJ2, J2);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M_Bot = pMesh_Bot[mesh_idx].pM->mcuvec();
				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M2_Bot = pMesh_Bot[mesh_idx].pM2->mcuvec();

				//relative coordinates to read value from bottom mesh (the one we're coupling to here) - relative to bottom mesh
				cuReal3 cell_rel_pos = cuReal3(
					(i + 0.5) * h.x + M.rect.s.x - M_Bot.rect.s.x,
					(j + 0.5) * h.y + M.rect.s.y - M_Bot.rect.s.y,
					M_Bot.rect.height() - M_Bot.h.z / 2);

				//can't couple to an empty cell
				if (!M_Bot.rect.contains(cell_rel_pos + M_Bot.rect.s) || M_Bot.is_empty(cell_rel_pos)) continue;

				//yes, then get value of magnetization used in coupling with current cell at cell_idx
				cuReal3 m_j1 = cu_normalize(M_Bot[cell_rel_pos]);
				cuReal3 m_j2 = cu_normalize(M2_Bot[cell_rel_pos]);
				cuReal3 m_i = cu_normalize(M[cell_idx]);

				//total surface exchange field in coupling cells, including bilinear and biquadratic terms
				cuReal3 Hsurfexch = (m_j1 / ((cuBReal)MU0 * Ms * h.z)) * J1 + (m_j2 / ((cuBReal)MU0 * Ms * h.z)) * J2;

				if (do_reduction) {

					energy_ = (-J1 * (m_i * m_j1) - J2 * (m_i * m_j2)) / (h.z * M.get_nonempty_cells());
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//Bottom mesh is atomistic
__global__ void SurfExchangeCUDA_BotAtom_UpdateField(ManagedMeshCUDA& cuMesh, ManagedAtom_MeshCUDA* pMesh_Bot, size_t coupled_meshes, ManagedModulesCUDA& cuModule, bool do_reduction)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC<cuReal3>& Heff = *cuMesh.pHeff;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	cuSZ3 n = M.n;
	cuReal3 h = M.h;

	cuBReal energy_ = 0.0;

	if (idx < n.x * n.y) {

		int i = idx % n.x;
		int j = idx / n.x;
		int cell_idx = i + j * n.x;

		//skip empty cells
		if (M.is_not_empty(cell_idx)) {

			cuBReal Ms = *cuMesh.pMs;
			cuMesh.update_parameters_mcoarse(cell_idx, *cuMesh.pMs, Ms);

			//check all meshes for coupling
			for (int mesh_idx = 0; mesh_idx < coupled_meshes; mesh_idx++) {

				mcuVEC_Managed<cuVEC_VC<cuReal3>, cuReal3>& M1 = pMesh_Bot[mesh_idx].pM1->mcuvec();

				//coupling rectangle in atomistic mesh in absolute coordinates
				cuRect rect_c = cuRect(
					cuReal3(i * h.x, j * h.y, M1.rect.e.z - M1.h.z),
					cuReal3((i + 1) * h.x, (j + 1) * h.y, M1.rect.e.z));
				rect_c += cuReal3(M.rect.s.x, M.rect.s.y, 0.0);

				//cells box in atomistic mesh
				cuBox acells = M1.box_from_rect_min(rect_c);

				//find total "directed energy" contribution from atomistic mesh : i.e. sum all mj * Js contributions from atomistic moments in the coupling area at the interface
				//NOTE : at atomistic/micromagnetic coupling, it's the atomistic mesh which sets coupling constant, not the top mesh
				cuReal3 total_directed_coupling_energy = cuReal3();
				for (int ai = acells.s.i; ai < acells.e.i; ai++) {
					for (int aj = acells.s.j; aj < acells.e.j; aj++) {

						cuReal3 rel_pos = cuReal3((ai + 0.5) * M1.h.x, (aj + 0.5) * M1.h.y, M1.rect.e.z - M1.h.z / 2);

						if (M1.is_empty(rel_pos)) continue;

						//Js value from atomistic mesh
						cuBReal Js = *pMesh_Bot[mesh_idx].pJs;
						cuBReal mu_s = *pMesh_Bot[mesh_idx].pmu_s;
						pMesh_Bot[mesh_idx].update_parameters_atposition(rel_pos, *pMesh_Bot[mesh_idx].pJs, Js, *pMesh_Bot[mesh_idx].pmu_s, mu_s);

						total_directed_coupling_energy += M1[rel_pos] * Js / mu_s;
					}
				}

				//now obtain coupling field from atomistic mesh at micromagnetic cell
				cuReal3 Hsurfexch = (total_directed_coupling_energy / (h.x * h.y)) / (MU0 * Ms * h.z);

				if (do_reduction) {

					energy_ = -MU0 * M[cell_idx] * Hsurfexch / M.get_nonempty_cells();
				}

				Heff[cell_idx] += Hsurfexch;

				if (do_reduction && cuModule.pModule_Heff->linear_size()) (*cuModule.pModule_Heff)[cell_idx] += Hsurfexch;
				if (do_reduction && cuModule.pModule_energy->linear_size()) (*cuModule.pModule_energy)[cell_idx] += energy_ * M.get_nonempty_cells();

				//for each cell, either it's not coupled to any other mesh cell (so we never get here), or else it's coupled to exactly one cell on this surface (thus can stop looping over meshes now)
				break;
			}
		}
	}

	if (do_reduction) reduction_sum(0, 1, &energy_, *cuModule.penergy);
}

//----------------------- UpdateField LAUNCHER

void SurfExchangeCUDA::UpdateField(void)
{
	if (pMeshCUDA->CurrentTimeStepSolved()) {

		ZeroEnergy();
		ZeroModuleVECs();

		//Coupling from ferromagnetic meshes

		//Top
		if (pMeshFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Coupling from antiferromagnetic meshes

		//Top
		if (pMeshAFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshAFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Coupling from atomistic meshes

		//Top
		if (pMeshAtom_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}

		//Bottom
		if (pMeshAtom_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), true);
			}
		}
	}
	else {

		//Coupling from ferromagnetic meshes

		//Top
		if (pMeshFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Coupling from antiferromagnetic meshes

		//Top
		if (pMeshAFM_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshAFM_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAFM_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Coupling from atomistic meshes

		//Top
		if (pMeshAtom_Top.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_TopAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}

		//Bottom
		if (pMeshAtom_Bot.size()) {

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				cuSZ3 dn = pMeshCUDA->M.device_n(mGPU);
				SurfExchangeCUDA_BotAtom_UpdateField <<< (dn.x * dn.y + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(pMeshCUDA->cuMesh.get_deviceobject(mGPU), pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), cuModule.get_deviceobject(mGPU), false);
			}
		}
	}
}

//----------------------- Initialization

//Current mesh is ferromagnetic
__global__ void set_SurfExchangeCUDA_pointers_kernel(
	ManagedMeshCUDA& cuMesh,
	ManagedMeshCUDA* pMeshFM_Bot, size_t coupledFM_bot_meshes,
	ManagedMeshCUDA* pMeshFM_Top, size_t coupledFM_top_meshes,
	ManagedMeshCUDA* pMeshAFM_Bot, size_t coupledAFM_bot_meshes,
	ManagedMeshCUDA* pMeshAFM_Top, size_t coupledAFM_top_meshes,
	ManagedAtom_MeshCUDA* pMeshAtom_Bot, size_t coupledAtom_bot_meshes,
	ManagedAtom_MeshCUDA* pMeshAtom_Top, size_t coupledAtom_top_meshes)
{
	if (threadIdx.x == 0) cuMesh.pMeshFM_Bot = pMeshFM_Bot;
	if (threadIdx.x == 1) cuMesh.pMeshFM_Bot_size = coupledFM_bot_meshes;
	if (threadIdx.x == 2) cuMesh.pMeshFM_Top = pMeshFM_Top;
	if (threadIdx.x == 3) cuMesh.pMeshFM_Top_size = coupledFM_top_meshes;

	if (threadIdx.x == 4) cuMesh.pMeshAFM_Bot = pMeshAFM_Bot;
	if (threadIdx.x == 5) cuMesh.pMeshAFM_Bot_size = coupledAFM_bot_meshes;
	if (threadIdx.x == 6) cuMesh.pMeshAFM_Top = pMeshAFM_Top;
	if (threadIdx.x == 7) cuMesh.pMeshAFM_Top_size = coupledAFM_top_meshes;

	if (threadIdx.x == 8) cuMesh.pMeshAtom_Bot = pMeshAtom_Bot;
	if (threadIdx.x == 9) cuMesh.pMeshAtom_Bot_size = coupledAtom_bot_meshes;
	if (threadIdx.x == 10) cuMesh.pMeshAtom_Top = pMeshAtom_Top;
	if (threadIdx.x == 11) cuMesh.pMeshAtom_Top_size = coupledAtom_top_meshes;
}

//Called by SurfExchangeCUDA module
void SurfExchangeCUDA::set_SurfExchangeCUDA_pointers(void)
{
	for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

		set_SurfExchangeCUDA_pointers_kernel <<< 1, CUDATHREADS >>>
			(pMeshCUDA->cuMesh.get_deviceobject(mGPU),
				pMeshFM_Bot.get_array(mGPU), pMeshFM_Bot.size(mGPU), pMeshFM_Top.get_array(mGPU), pMeshFM_Top.size(mGPU),
				pMeshAFM_Bot.get_array(mGPU), pMeshAFM_Bot.size(mGPU), pMeshAFM_Top.get_array(mGPU), pMeshAFM_Top.size(mGPU),
				pMeshAtom_Bot.get_array(mGPU), pMeshAtom_Bot.size(mGPU), pMeshAtom_Top.get_array(mGPU), pMeshAtom_Top.size(mGPU));
	}
}

#endif

#endif

