#include "hip/hip_runtime.h"
#include "Mesh_FerromagneticCUDA.h"

#if COMPILECUDA == 1

#ifdef MESH_COMPILATION_FERROMAGNETIC

#include "BorisCUDALib.cuh"

#include "MeshCUDA.h"
#include "MeshParamsControlCUDA.h"

__global__ void Zero_MCAux_Atom_Mesh_FMCUDA(cuBReal& aux_real)
{
	if (threadIdx.x == 0) aux_real = 0.0;
}

///////////////////////////////////////////////////////////////
// PARALLEL MONTE-CARLO METROPOLIS - WITH REDUCTION

__global__ void Iterate_MonteCarloCUDA_Classic_FM_red_kernel(ManagedMeshCUDA& cuMesh, int* cuModules, int& numModules, cuReal3& Ha, cuBorisRand<>& prng, cuBReal mc_cone_angledeg, cuBReal& mc_acceptance_rate)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;

	cuSZ3& n = M.n;

	int num_moves = M.get_nonempty_cells();

	cuBReal acceptance_rate = 0.0;

	//this method must be called with half-size : n.dim() / 2, i.e. <<< (n.dim() / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
	//double idx : idx values will now take on even values
	int spin_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

	//ijk coordinates corresponding to idx
	cuINT3 ijk = cuINT3(spin_idx % n.x, (spin_idx / n.x) % n.y, spin_idx / (n.x*n.y));

	if (n.x % 2 == 1) {
		if (n.y % 2 == 0) {

			//nx is odd and ny is even : for red squares nudge on odd planes only
			spin_idx += (int)(n.z % 2);
		}
		//else : nx is odd and ny is odd, no nudge is needed
	}
	else {

		//nx is even : for red squares nudge on a) odd rows and even planes, and b) even rows and odd planes

		//red_nudge = true for odd rows and even planes or for even rows and odd planes - have to keep index on the checkerboard pattern
		bool red_nudge = (((ijk.j % 2) == 1 && (ijk.k % 2) == 0) || (((ijk.j % 2) == 0 && (ijk.k % 2) == 1)));

		spin_idx += (int)red_nudge;
	}

	//calculate only in non-empty and non-frozen cells
	if (spin_idx < n.dim() && M.is_not_empty(spin_idx) && !M.is_skipcell(spin_idx)) {

		cuBReal Ms_val = *cuMesh.pMs;
		cuBReal susrel_val = *cuMesh.psusrel;
		cuMesh.update_parameters_mcoarse(spin_idx, *cuMesh.pMs, Ms_val, *cuMesh.psusrel, susrel_val);

		cuBReal Temperature;
		if (Temp.linear_size()) Temperature = Temp[M.cellidx_to_position(spin_idx)];
		else Temperature = *cuMesh.pbase_temperature;

		cuBReal Ms0 = cuMesh.pMs->get0();
		cuBReal me = Ms_val / Ms0;

		//Picked spin is M[spin_idx]
		cuReal3 M_old = M[spin_idx];

		//obtain rotated spin in a cone around the picked spin
		cuBReal theta_rot = prng.rand() * mc_cone_angledeg * PI / 180.0;
		cuBReal phi_rot = prng.rand() * 2 * PI;
		//Move spin in cone with uniform random probability distribution.
		cuReal3 M_new = relrotate_polar(M_old, theta_rot, phi_rot);

		//now allow magnetization length to change slightly with a Gaussian pdf around current value with sigma value from the normal distribution of P(m^2).
		if (Temperature > 0.0) {

			cuBReal sigma = 2 * me*sqrt(susrel_val*(cuBReal)BOLTZMANN*Temperature / (M.h.dim() * Ms0));
			if (Temperature >= *cuMesh.pT_Curie || sigma > 0.03) sigma = 0.03;
			M_new *= 1 + (prng.rand() * 2 * sigma - sigma);
		}

		//1. Find energy change
		cuBReal energy_delta = cuMesh.Get_EnergyChange_FM(spin_idx, M_new, cuModules, numModules, Ha);

		//2. Find contribution to free energy change from longitudinal susceptibility
		cuReal3 m = M_old / Ms0;
		cuReal3 m_new = M_new / Ms0;

		if (Temperature > 0.0 && Temperature <= *cuMesh.pT_Curie) {

			cuBReal diff = m * m - me * me;
			cuBReal diff_new = m_new * m_new - me * me;

			energy_delta += M.h.dim() * (Ms0 / (8 * susrel_val * me*me)) * (diff_new * diff_new - diff * diff);
		}
		else if (Temperature > 0.0) {

			cuBReal r = 3 * *cuMesh.pT_Curie / (10 * (Temperature - *cuMesh.pT_Curie));
			cuBReal m_new_sq = m_new * m_new;
			cuBReal m_sq = m * m;
			energy_delta += M.h.dim() * (Ms0 / (2 * susrel_val)) * (m_new_sq * (1 + r * m_new_sq) - m_sq * (1 + r * m_sq));
		}

		//Compute acceptance probability.
		//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
		cuBReal P_accept = 0.0, P = 1.0;
		if (Temperature > 0.0) {

			//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
			cuBReal Mratio = (M_new*M_new) / (M_old*M_old);
			P_accept = Mratio * Mratio * exp(-energy_delta / ((cuBReal)BOLTZMANN * Temperature));
			//uniform random number between 0 and 1
			P = prng.rand();
		}
		else if (energy_delta < 0) P_accept = 1.0;
		
		if (P <= P_accept) {

			acceptance_rate = 1.0 / num_moves;

			//set new spin
			M[spin_idx] = M_new;
		}
	}

	reduction_sum(0, 1, &acceptance_rate, mc_acceptance_rate);
}

__global__ void Iterate_MonteCarloCUDA_Classic_FM_black_kernel(ManagedMeshCUDA& cuMesh, int* cuModules, int& numModules, cuReal3& Ha, cuBorisRand<>& prng, cuBReal mc_cone_angledeg, cuBReal& mc_acceptance_rate)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;

	cuSZ3& n = M.n;

	int num_moves = M.get_nonempty_cells();

	cuBReal acceptance_rate = 0.0;

	//this method must be called with half-size : n.dim() / 2, i.e. <<< (n.dim() / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
	//double idx : idx values will now take on even values
	int spin_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

	//ijk coordinates corresponding to idx
	cuINT3 ijk = cuINT3(spin_idx % n.x, (spin_idx / n.x) % n.y, spin_idx / (n.x*n.y));

	if (n.x % 2 == 1) {
		if (n.y % 2 == 0) {

			//nx is odd and ny is even : for black squares nudge on even planes only
			spin_idx += (int)(n.z % 2 == 0);
		}
		else {

			//nx is odd and ny is odd, nudge everything by 1 for black squares
			spin_idx++;
		}
	}
	else {

		//nx is even : for black squares nudge on a) even rows and even planes, and b) odd rows and odd planes

		//red_nudge = true for odd rows and even planes or for even rows and odd planes - have to keep index on the checkerboard pattern
		bool black_nudge = (((ijk.j % 2) == 0 && (ijk.k % 2) == 0) || (((ijk.j % 2) == 1 && (ijk.k % 2) == 1)));

		spin_idx += (int)black_nudge;
	}

	//calculate only in non-empty and non-frozen cells
	if (spin_idx < n.dim() && M.is_not_empty(spin_idx) && !M.is_skipcell(spin_idx)) {

		cuBReal Ms_val = *cuMesh.pMs;
		cuBReal susrel_val = *cuMesh.psusrel;
		cuMesh.update_parameters_mcoarse(spin_idx, *cuMesh.pMs, Ms_val, *cuMesh.psusrel, susrel_val);

		cuBReal Temperature;
		if (Temp.linear_size()) Temperature = Temp[M.cellidx_to_position(spin_idx)];
		else Temperature = *cuMesh.pbase_temperature;

		cuBReal Ms0 = cuMesh.pMs->get0();
		cuBReal me = Ms_val / Ms0;

		//Picked spin is M[spin_idx]
		cuReal3 M_old = M[spin_idx];

		//obtain rotated spin in a cone around the picked spin
		cuBReal theta_rot = prng.rand() * mc_cone_angledeg * PI / 180.0;
		cuBReal phi_rot = prng.rand() * 2 * PI;
		//Move spin in cone with uniform random probability distribution.
		cuReal3 M_new = relrotate_polar(M_old, theta_rot, phi_rot);

		//now allow magnetization length to change slightly with a Gaussian pdf around current value with sigma value from the normal distribution of P(m^2).
		if (Temperature > 0.0) {

			cuBReal sigma = 2 * me*sqrt(susrel_val*(cuBReal)BOLTZMANN*Temperature / (M.h.dim() * Ms0));
			if (Temperature >= *cuMesh.pT_Curie || sigma > 0.03) sigma = 0.03;
			M_new *= 1 + (prng.rand() * 2 * sigma - sigma);
		}

		//1. Find energy change
		cuBReal energy_delta = cuMesh.Get_EnergyChange_FM(spin_idx, M_new, cuModules, numModules, Ha);

		//2. Find contribution to free energy change from longitudinal susceptibility
		cuReal3 m = M_old / Ms0;
		cuReal3 m_new = M_new / Ms0;

		if (Temperature > 0.0 && Temperature <= *cuMesh.pT_Curie) {

			cuBReal diff = m * m - me * me;
			cuBReal diff_new = m_new * m_new - me * me;

			energy_delta += M.h.dim() * (Ms0 / (8 * susrel_val * me*me)) * (diff_new * diff_new - diff * diff);
		}
		else if (Temperature > 0.0) {

			cuBReal r = 3 * *cuMesh.pT_Curie / (10 * (Temperature - *cuMesh.pT_Curie));
			cuBReal m_new_sq = m_new * m_new;
			cuBReal m_sq = m * m;
			energy_delta += M.h.dim() * (Ms0 / (2 * susrel_val)) * (m_new_sq * (1 + r * m_new_sq) - m_sq * (1 + r * m_sq));
		}

		//Compute acceptance probability.
		//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
		cuBReal P_accept = 0.0, P = 1.0;
		if (Temperature > 0.0) {

			//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
			cuBReal Mratio = (M_new*M_new) / (M_old*M_old);
			P_accept = Mratio * Mratio * exp(-energy_delta / ((cuBReal)BOLTZMANN * Temperature));
			//uniform random number between 0 and 1
			P = prng.rand();
		}
		else if (energy_delta < 0) P_accept = 1.0;

		if (P <= P_accept) {

			acceptance_rate = 1.0 / num_moves;

			//set new spin
			M[spin_idx] = M_new;
		}
	}

	reduction_sum(0, 1, &acceptance_rate, mc_acceptance_rate);
}

///////////////////////////////////////////////////////////////
// PARALLEL MONTE-CARLO METROPOLIS - WITHOUT REDUCTION

__global__ void Iterate_MonteCarloCUDA_Classic_FM_red_kernel(ManagedMeshCUDA& cuMesh, int* cuModules, int& numModules, cuReal3& Ha, cuBorisRand<>& prng, cuBReal mc_cone_angledeg)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;

	cuSZ3& n = M.n;

	int num_moves = M.get_nonempty_cells();

	//this method must be called with half-size : n.dim() / 2, i.e. <<< (n.dim() / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
	//double idx : idx values will now take on even values
	int spin_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

	//ijk coordinates corresponding to idx
	cuINT3 ijk = cuINT3(spin_idx % n.x, (spin_idx / n.x) % n.y, spin_idx / (n.x*n.y));

	if (n.x % 2 == 1) {
		if (n.y % 2 == 0) {

			//nx is odd and ny is even : for red squares nudge on odd planes only
			spin_idx += (int)(n.z % 2);
		}
		//else : nx is odd and ny is odd, no nudge is needed
	}
	else {

		//nx is even : for red squares nudge on a) odd rows and even planes, and b) even rows and odd planes

		//red_nudge = true for odd rows and even planes or for even rows and odd planes - have to keep index on the checkerboard pattern
		bool red_nudge = (((ijk.j % 2) == 1 && (ijk.k % 2) == 0) || (((ijk.j % 2) == 0 && (ijk.k % 2) == 1)));

		spin_idx += (int)red_nudge;
	}

	//calculate only in non-empty and non-frozen cells
	if (spin_idx < n.dim() && M.is_not_empty(spin_idx) && !M.is_skipcell(spin_idx)) {

		cuBReal Ms_val = *cuMesh.pMs;
		cuBReal susrel_val = *cuMesh.psusrel;
		cuMesh.update_parameters_mcoarse(spin_idx, *cuMesh.pMs, Ms_val, *cuMesh.psusrel, susrel_val);

		cuBReal Temperature;
		if (Temp.linear_size()) Temperature = Temp[M.cellidx_to_position(spin_idx)];
		else Temperature = *cuMesh.pbase_temperature;

		cuBReal Ms0 = cuMesh.pMs->get0();
		cuBReal me = Ms_val / Ms0;

		//Picked spin is M[spin_idx]
		cuReal3 M_old = M[spin_idx];

		//obtain rotated spin in a cone around the picked spin
		cuBReal theta_rot = prng.rand() * mc_cone_angledeg * PI / 180.0;
		cuBReal phi_rot = prng.rand() * 2 * PI;
		//Move spin in cone with uniform random probability distribution.
		cuReal3 M_new = relrotate_polar(M_old, theta_rot, phi_rot);

		//now allow magnetization length to change slightly with a Gaussian pdf around current value with sigma value from the normal distribution of P(m^2).
		if (Temperature > 0.0) {

			cuBReal sigma = 2 * me*sqrt(susrel_val*(cuBReal)BOLTZMANN*Temperature / (M.h.dim() * Ms0));
			if (Temperature >= *cuMesh.pT_Curie || sigma > 0.03) sigma = 0.03;
			M_new *= 1 + (prng.rand() * 2 * sigma - sigma);
		}

		//1. Find energy change
		cuBReal energy_delta = cuMesh.Get_EnergyChange_FM(spin_idx, M_new, cuModules, numModules, Ha);

		//2. Find contribution to free energy change from longitudinal susceptibility
		cuReal3 m = M_old / Ms0;
		cuReal3 m_new = M_new / Ms0;

		if (Temperature > 0.0 && Temperature <= *cuMesh.pT_Curie) {

			cuBReal diff = m * m - me * me;
			cuBReal diff_new = m_new * m_new - me * me;

			energy_delta += M.h.dim() * (Ms0 / (8 * susrel_val * me*me)) * (diff_new * diff_new - diff * diff);
		}
		else if (Temperature > 0.0) {

			cuBReal r = 3 * *cuMesh.pT_Curie / (10 * (Temperature - *cuMesh.pT_Curie));
			cuBReal m_new_sq = m_new * m_new;
			cuBReal m_sq = m * m;
			energy_delta += M.h.dim() * (Ms0 / (2 * susrel_val)) * (m_new_sq * (1 + r * m_new_sq) - m_sq * (1 + r * m_sq));
		}

		//Compute acceptance probability.
		//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
		cuBReal P_accept = 0.0, P = 1.0;
		if (Temperature > 0.0) {

			//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
			cuBReal Mratio = (M_new*M_new) / (M_old*M_old);
			P_accept = Mratio * Mratio * exp(-energy_delta / ((cuBReal)BOLTZMANN * Temperature));
			//uniform random number between 0 and 1
			P = prng.rand();
		}
		else if (energy_delta < 0) P_accept = 1.0;

		if (P <= P_accept) {

			//set new spin
			M[spin_idx] = M_new;
		}
	}
}

__global__ void Iterate_MonteCarloCUDA_Classic_FM_black_kernel(ManagedMeshCUDA& cuMesh, int* cuModules, int& numModules, cuReal3& Ha, cuBorisRand<>& prng, cuBReal mc_cone_angledeg)
{
	cuVEC_VC<cuReal3>& M = *cuMesh.pM;
	cuVEC_VC<cuBReal>& Temp = *cuMesh.pTemp;

	cuSZ3& n = M.n;

	int num_moves = M.get_nonempty_cells();

	//this method must be called with half-size : n.dim() / 2, i.e. <<< (n.dim() / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
	//double idx : idx values will now take on even values
	int spin_idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

	//ijk coordinates corresponding to idx
	cuINT3 ijk = cuINT3(spin_idx % n.x, (spin_idx / n.x) % n.y, spin_idx / (n.x*n.y));

	if (n.x % 2 == 1) {
		if (n.y % 2 == 0) {

			//nx is odd and ny is even : for black squares nudge on even planes only
			spin_idx += (int)(n.z % 2 == 0);
		}
		else {

			//nx is odd and ny is odd, nudge everything by 1 for black squares
			spin_idx++;
		}
	}
	else {

		//nx is even : for black squares nudge on a) even rows and even planes, and b) odd rows and odd planes

		//red_nudge = true for odd rows and even planes or for even rows and odd planes - have to keep index on the checkerboard pattern
		bool black_nudge = (((ijk.j % 2) == 0 && (ijk.k % 2) == 0) || (((ijk.j % 2) == 1 && (ijk.k % 2) == 1)));

		spin_idx += (int)black_nudge;
	}

	//calculate only in non-empty and non-frozen cells
	if (spin_idx < n.dim() && M.is_not_empty(spin_idx) && !M.is_skipcell(spin_idx)) {

		cuBReal Ms_val = *cuMesh.pMs;
		cuBReal susrel_val = *cuMesh.psusrel;
		cuMesh.update_parameters_mcoarse(spin_idx, *cuMesh.pMs, Ms_val, *cuMesh.psusrel, susrel_val);

		cuBReal Temperature;
		if (Temp.linear_size()) Temperature = Temp[M.cellidx_to_position(spin_idx)];
		else Temperature = *cuMesh.pbase_temperature;

		cuBReal Ms0 = cuMesh.pMs->get0();
		cuBReal me = Ms_val / Ms0;

		//Picked spin is M[spin_idx]
		cuReal3 M_old = M[spin_idx];

		//obtain rotated spin in a cone around the picked spin
		cuBReal theta_rot = prng.rand() * mc_cone_angledeg * PI / 180.0;
		cuBReal phi_rot = prng.rand() * 2 * PI;
		//Move spin in cone with uniform random probability distribution.
		cuReal3 M_new = relrotate_polar(M_old, theta_rot, phi_rot);

		//now allow magnetization length to change slightly with a Gaussian pdf around current value with sigma value from the normal distribution of P(m^2).
		if (Temperature > 0.0) {

			cuBReal sigma = 2 * me*sqrt(susrel_val*(cuBReal)BOLTZMANN*Temperature / (M.h.dim() * Ms0));
			if (Temperature >= *cuMesh.pT_Curie || sigma > 0.03) sigma = 0.03;
			M_new *= 1 + (prng.rand() * 2 * sigma - sigma);
		}

		//1. Find energy change
		cuBReal energy_delta = cuMesh.Get_EnergyChange_FM(spin_idx, M_new, cuModules, numModules, Ha);

		//2. Find contribution to free energy change from longitudinal susceptibility
		cuReal3 m = M_old / Ms0;
		cuReal3 m_new = M_new / Ms0;

		if (Temperature > 0.0 && Temperature <= *cuMesh.pT_Curie) {

			cuBReal diff = m * m - me * me;
			cuBReal diff_new = m_new * m_new - me * me;

			energy_delta += M.h.dim() * (Ms0 / (8 * susrel_val * me*me)) * (diff_new * diff_new - diff * diff);
		}
		else if (Temperature > 0.0) {

			cuBReal r = 3 * *cuMesh.pT_Curie / (10 * (Temperature - *cuMesh.pT_Curie));
			cuBReal m_new_sq = m_new * m_new;
			cuBReal m_sq = m * m;
			energy_delta += M.h.dim() * (Ms0 / (2 * susrel_val)) * (m_new_sq * (1 + r * m_new_sq) - m_sq * (1 + r * m_sq));
		}

		//Compute acceptance probability.
		//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
		cuBReal P_accept = 0.0, P = 1.0;
		if (Temperature > 0.0) {

			//Target pdf is proportional to M^2 * exp(-E/kBT), however spin picking probability is not uniform, but proportional to M^2. Thus acceptance probability required to satisfy detailed balance is min{1, (M_new^4 / M_old^4) * exp(-dE/kBT)}
			cuBReal Mratio = (M_new*M_new) / (M_old*M_old);
			P_accept = Mratio * Mratio * exp(-energy_delta / ((cuBReal)BOLTZMANN * Temperature));
			//uniform random number between 0 and 1
			P = prng.rand();
		}
		else if (energy_delta < 0) P_accept = 1.0;

		if (P <= P_accept) {

			//set new spin
			M[spin_idx] = M_new;
		}
	}
}

cuBReal FMeshCUDA::Iterate_MonteCarloCUDA_Classic(cuBReal mc_cone_angledeg, double target_acceptance_rate)
{
	if (mc_acceptance_reduction_counter == 0) {

		for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

			Zero_MCAux_Atom_Mesh_FMCUDA <<< 1, CUDATHREADS >>> (mc_acceptance_rate(mGPU));
		}
	}

	//Field set
	if (pHa) {

		if (mc_acceptance_reduction_counter == 0) {

			//with acceptance rate reduction
			
			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_red_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), (*pHa)(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg, mc_acceptance_rate(mGPU));
			}

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_black_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), (*pHa)(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg, mc_acceptance_rate(mGPU));
			}
		}
		else {

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				//without acceptance rate reduction
				Iterate_MonteCarloCUDA_Classic_FM_red_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), (*pHa)(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg);
			}

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_black_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), (*pHa)(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg);
			}
		}
	}
	//No field (or rather ZeemanCUDA module not added)
	else {

		mcu_val<cuReal3> Ha(mGPU);
		Ha.from_cpu(cuReal3());

		if (mc_acceptance_reduction_counter == 0) {

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				//with acceptance rate reduction
				Iterate_MonteCarloCUDA_Classic_FM_red_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), Ha(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg, mc_acceptance_rate(mGPU));
			}

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_black_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), Ha(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg, mc_acceptance_rate(mGPU));
			}
		}
		else {

			//without acceptance rate reduction
			
			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_red_kernel << < (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >> >
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), Ha(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg);
			}

			//must refresh halos before both red and black passes
			M.exchange_halos();

			for (mGPU.device_begin(); mGPU != mGPU.device_end(); mGPU++) {

				Iterate_MonteCarloCUDA_Classic_FM_black_kernel <<< (M.device_size(mGPU) / 2 + CUDATHREADS) / CUDATHREADS, CUDATHREADS >>>
					(cuMesh.get_deviceobject(mGPU), cuModules(mGPU), cuNumModules(mGPU), Ha(mGPU), prng.get_deviceobject(mGPU), mc_cone_angledeg);
			}
		}
	}

	if (mc_acceptance_reduction_counter == 0) {

		mc_acceptance_rate_last = mc_acceptance_rate.to_cpu_sum() / mGPU.get_num_devices();

		//is acceptance rate close enough to target acceptance? If yes don't do reduction next time.
		if (abs(target_acceptance_rate - mc_acceptance_rate_last) < MONTECARLO_ACCEPTANCETOLERANCE) mc_acceptance_reduction_counter = 1;

		return mc_acceptance_rate_last;
	}
	else {

		//increase counter until we come full circle : when mc_acceptance_reduction_counter becomes zero again we'll have to do reduction just to check.
		mc_acceptance_reduction_counter = (mc_acceptance_reduction_counter + 1) % MONTECARLO_REDUCTIONITERS;

		//return exact terget acceptance rate means cone angle will not be adjusted
		return target_acceptance_rate;
	}
}

#endif

#endif